#include "hip/hip_runtime.h"
/*********************************************************/
/*                                                       */
/*                     k-means.c                         */
/*                     =========                         */
/*                                                       */
/*        C programme to demonstrate k-means             */
/*                clustering on 2D data                  */
/*                                                       */
/*          Written for EE3J2 Data Mining                */
/*                                                       */
/*        Version 1: Martin Russell 26/02/04             */
/*                                                       */
/* Dept. Electronic, Electrical & Computer Engineering   */
/*            University of Birmingham                   */
/*                                                       */
/*    To compile under linux:                            */
/*                 gcc -lm k-means.c                     */
/*                 mv a.out k-means                      */
/*                                                       */
/*    To run:                                            */
/*                 k-means ipFile centroids opFile numIt */
/*                                                       */
/*                                                       */
/*                  CUDA COMPATIBILITY                   */
/*                  ==================                   */
/*                                                       */
/*                                                       */
/*  Changed by:                                          */
/*    Rui Albuquerque                                    */
/*    Artur Gon�alves                                    */
/*    Daniel Filipe                                      */
/*                                                       */
/*    for: Advanced Computer Architectures Class         */
/*        Electrical and Computer Engineering Department */
/*        Instituto Superior T�cnico                     */
/*        Spring Semester                                */
/*        2013/1014                                      */
/*                                                       */
/*  To run:                                              */
/*  kmeans numPoints numCents Dims Range RandSeed <numIt>*/
/*                                                       */
/*                                                       */
/*********************************************************/

#include <stdio.h>
#include <stdlib.h>
#include <limits.h>
#include <math.h>
#include <string.h>
#include <math.h>
#include <ctype.h>
#include <time.h>

#include "hip/hip_runtime.h"

#include "kernel.h"

#ifdef COUNTTIME
/* Calculates the difference between two times, in seconds. */
double timeDiff(struct timespec tStart, struct timespec tEnd);
#endif

void mySrand(int seed);
int myRand();

int main(int argc, char *argv[])
{

	int j;
	int k;
	int n;

	/*Host data*/
	data_t *x;		/* Max values: x[numPoints * Dimensions]. Coordinate of point P, all 'dims' of them. Indexed as [P * dims + dim]. */
	data_t *cent;	/* Coordinates of each centroid. Max value: [numCent * dims]. Layout is [cx1 cy1 cx2 cy2 cx3 cy3 ...] in case dims=2  */
	data_t *mean;	/* Length: dims */
	data_t *dist;	/* Distance between Point P and Centroid C. Indexed as [P][C]. */
	int *bestCent;	/* Length: numPoints */
	data_t *cent_r; /* Length: dims. Used as temporary variable. */

	/*Device data*/
	/* TODO: put device_x in constant texture memory - should be faster */
	data_t *device_x;		/* Max value: [numPoints * dims]. Layout is [x1 y1 x2 y2 x3 y3 ...] in case dims=2 */
	data_t *device_cent;	/* Max value: [numCent * dims]. Layout is [cx1 cy1 cx2 cy2 cx3 cy3 ...] in case dims=2 */
	data_t *device_mean;	/* Max value: [dims]. */
	data_t *device_dist;	/* Max value: [numPoints * numCent]. Distance between Point P and Centroid C. Indexed as [P * numCent + C]. */
	int *device_bestCent;	/* Length: numPoints */
	data_t *device_cent_r;	/* Length: dims * numCent. Used as temporary variables for accumulation. */
	int *device_cent_tot;	/* Number of points for each centroid, global to all blocks. Length: numCent */
	int *device_cent_partial_tot;	/* Number of points for each centroid, local to one block. Length: numCent */

	int numPoints;	/* Number of points */
	int dims;		/* Dimensions */
	int numCent;	/* Number of centroids */
	int NUMIT = 100;

	int range;
	int randSeed;

	int numThreads;	/* Number of threads per block */
	int numBlocks;	/* Number of blocks */

	void * hostErr [6];  // Errors for malloc
	hipError_t err[8]; // Errors for hipMalloc and hipMemcpy

#ifdef COUNTTIME
	struct timespec hostStartTime;
	struct timespec hostEndTime;

	struct timespec startTime;
	struct timespec endTime;

	struct timespec startCommTime;
	struct timespec endCommTime;

	struct timespec memBackStartTime;
	struct timespec memBackEndTime;
#endif

	/* Check correct number of input parameters */
	if ((argc!=6)&&(argc!=7))
	{
		printf("format: k-means numPoints numCents Dims Range RandSeed <numIt>\n");
		exit(1);
	}

	printf("Reading Arguments\n");

	numPoints	= atoi(argv[1]);
	numCent		= atoi(argv[2]);
	dims		= atoi(argv[3]);
	range		= atoi(argv[4]);
	randSeed	= atoi(argv[5]);

	if(argc == 7)
		NUMIT = atoi(argv[6]);

	if(numCent > numPoints)
	{
		printf("Number of Centroids must be smaller than the number of points");
		exit(1);
	}

	printf("Starting with arguments:\n\
		   numPoints = %d\n\
		   numCents  = %d\n\
		   Dims      = %d\n\
		   Range     = %d\n\
		   RandSeed  = %d\n\
		   numIt     = %d\n\n",
		   numPoints, numCent, dims, range, randSeed, NUMIT);

	printf("Allocating host memory\n");

	/* allocate memory */
	hostErr[0] = x = (data_t *)calloc(numPoints * dims,sizeof(data_t *));
	hostErr[1] = cent=(data_t *)calloc(numCent * dims,sizeof(data_t *));
	hostErr[2] = mean=(data_t *)calloc(dims,sizeof(data_t));
	hostErr[3] = dist=(data_t *)calloc(numPoints * numCent,sizeof(data_t *));
	hostErr[4] = bestCent=(int *)calloc(numPoints,sizeof(int));
	hostErr[5] = cent_r=(data_t *)calloc(dims,sizeof(data_t));
	
	for(n = 0; n < 6; n++) {
		if(hostErr[n] == NULL){
			printf("Error allocating memory on host (error code %s). Exiting.", hipGetErrorString(err[n]));
			exit(0);
		}
	}

	/* Calculate kernel parameters */
	numThreads = numPoints;
	numBlocks = 1;


	/* Memory allocation on the CUDA device */

	/* Touch the device once to initialize it */
	err[0] = hipFree(0);
	if (err[0] != hipSuccess)
	{
		printf("Error on first touch (hipFree(0)): %s\nExiting.\n", hipGetErrorString(err[0]));
		exit(1);
	}

	/*Allocate device memory*/
	printf("Allocating device memory\n");

	err[0] = hipMalloc ((void **) &device_x, numPoints * dims * sizeof(data_t));
	err[1] = hipMalloc ((void **) &device_cent, numCent * dims * sizeof(data_t));
	err[2] = hipMalloc ((void **) &device_mean, dims * sizeof(data_t));
	err[3] = hipMalloc ((void **) &device_dist, numPoints * numCent * sizeof(data_t));
	err[4] = hipMalloc ((void **) &device_bestCent, numPoints * sizeof(int));
	err[5] = hipMalloc ((void **) &device_cent_r, dims * numCent * sizeof(data_t));
	
	err[6] = hipMalloc ((void **) &device_cent_tot, numCent * sizeof(int));
	err[7] = hipMalloc ((void **) &device_cent_partial_tot, numCent * numBlocks * sizeof(int));

	for(n = 0; n < 8; n++) {
		if(err[n] != hipSuccess){
			printf("Error allocating memory on device: %s\nExiting.\n", hipGetErrorString(err[n]));
			exit(1);
		}
	}

	/*Generate random test set according to user specification*/
	printf("Generating Test Set\n");

	mySrand(randSeed);

	for(n = 0; n < numPoints; n ++)
	{
		for(j = 0; j < dims; j++)
		{
			X_(n,j) = myRand() % range;
			mean[j] += X_(n,j);

			if(n < numCent){
				CENT_(n, j) = myRand() % range;
			}

		}
	}

	for(n = 0; n < dims; n++)
	{
		mean[n] = mean[n]/numPoints;
	}

	/* Sending Data to Device*/

#ifdef COUNTTIME
	clock_gettime(CLOCK_REALTIME, &startCommTime);
#endif

	err[0] = hipMemcpy (device_x, x, numPoints * dims * sizeof(data_t), hipMemcpyHostToDevice);
	err[1] = hipMemcpy (device_cent, cent, numCent * dims * sizeof(data_t), hipMemcpyHostToDevice);
	err[2] = hipMemcpy (device_mean, mean, dims * sizeof(data_t), hipMemcpyHostToDevice);
	err[3] = hipMemcpy (device_dist, dist, numPoints * numCent * sizeof(data_t), hipMemcpyHostToDevice);
	err[4] = hipMemcpy (device_bestCent,bestCent, numPoints * sizeof(int), hipMemcpyHostToDevice);
	err[5] = hipMemcpy (device_cent_r, cent_r,dims * sizeof(data_t), hipMemcpyHostToDevice);

	err[6] = hipMemset (device_cent_tot, 0, numCent * sizeof(int));
	err[7] = hipMemset (device_cent_partial_tot, 0, numCent * numBlocks * sizeof(int));

	for(n = 0; n < 8; n++) {
		if(err[n] != hipSuccess){
			printf("Error allocating memory on device (error code %s). Exiting.", hipGetErrorString(err[n]));
			exit(0);
		}
	}

#ifdef COUNTTIME
	clock_gettime(CLOCK_REALTIME, &endCommTime);
#endif

	printf("Starting host calculation.\n");

#ifdef COUNTTIME
	clock_gettime(CLOCK_REALTIME, &hostStartTime);
#endif
	
	// Run algorithm on host for correctness check
	hostKmeans(NUMIT, numPoints, numCent, dims,
			x, dist, cent,
			mean, bestCent, cent_r);

#ifdef COUNTTIME
	clock_gettime(CLOCK_REALTIME, &hostEndTime);
#endif

	printf("Starting device calculation.\n");

	size_t heap_size;
	hipDeviceGetLimit(&heap_size, hipLimitMallocHeapSize);

#ifdef COUNTTIME
	clock_gettime(CLOCK_REALTIME, &startTime);
#endif

	kernel<<<numBlocks, numThreads>>>(NUMIT, numPoints, numCent, dims,
					device_x, device_dist, device_cent,
					device_mean, device_bestCent, device_cent_r,
					1,
					device_cent_partial_tot,
					device_cent_tot
					);

	hipDeviceSynchronize();

	err[0] = hipGetLastError();
	if (err[0] != hipSuccess)
	{
		printf("Oh shit, shit happened: %s\n", hipGetErrorString(err[0]));
	}

#ifdef COUNTTIME
	clock_gettime(CLOCK_REALTIME, &endTime);
#endif

	printf("End of device calculation.\n");

	/* Pull results back from device */
	int *cudaBestCent = (int *) calloc(numPoints, sizeof(int));
	if (cudaBestCent == NULL)
	{
		printf("Failed to allocate extra space for device's result.\n");
		exit(1);
	}

	data_t *cudaCent = (data_t *) calloc(numCent * dims, sizeof(data_t));
	if (cudaCent == NULL)
	{
		printf("Failed to allocate extra space for device's result.\n");
		exit(1);
	}

#ifdef COUNTTIME
	clock_gettime(CLOCK_REALTIME, &memBackStartTime);
#endif

	err[0] = hipMemcpy (cudaBestCent, device_bestCent, numPoints * sizeof(int), hipMemcpyDeviceToHost);
	err[1] = hipMemcpy (cudaCent, device_cent, numCent * dims * sizeof(int), hipMemcpyDeviceToHost);
	
	if (err[0] != hipSuccess || err[1] != hipSuccess)
	{
		printf("Failed to transfer device's result: %s\n", hipGetErrorString(err[0]));
		exit(1);
	}

#ifdef COUNTTIME
	clock_gettime(CLOCK_REALTIME, &memBackEndTime);
#endif

	/* Verify device results */
	for (j = 0; j < numPoints; j++)
	{
		if (cudaBestCent[j] != bestCent[j])
		{
			printf("Error: Host and device bestCent results do not match.\n");
			printf("Error at %d\n\tHost has %d\n\tDevice has %d\n", j, bestCent[j], cudaBestCent[j]);
			exit(1);
		}
	}

	for (k = 0; k < numCent; k++)
	{
		int e;
		for (e = 0; e < dims; e++)
		{
			if (cent[k * dims + e] != cudaCent[k * dims + e])
			{
				printf("Error: Host and device cent results do not match.\n");
				printf("Error at centroid %d\n", k);
				exit(1);
			}
		}
	}

	/* write clusters to screen */
	printf("\nDevice results\n=========\n");

	for (k=0; k<numCent; k++)
	{
		printf("\nCluster %d\n=========\n",k);
		for (j=0; j<numPoints; j++)
		{
			if (cudaBestCent[j]==k)
				printf("point %d\n",j);
		}
	}

#ifdef COUNTTIME

	double hostTime = timeDiff(hostStartTime, hostEndTime);
	double deviceTime = timeDiff(startTime, endTime);

	printf("\nTime Report\n=======\n\n");
	printf("Communication Host->Device time: %f s\n", timeDiff(startCommTime, endCommTime));
	printf("Algorithm Host Computation:      %f s\n", hostTime);
	printf("Algorithm Device Computation:    %f s\n", deviceTime);
	printf("Communication Device->Host time: %f s\n", timeDiff(memBackStartTime, memBackEndTime));
	printf("Speed-up: %f\n", hostTime / deviceTime);
#endif

	exit(0);
}

#ifdef COUNTTIME
/**
* timeDiff
*
* Computes the difference (in ns) between the start and end time
*/
double timeDiff(struct timespec tStart, struct timespec tEnd)
{
	struct timespec diff;

	diff.tv_sec  = tEnd.tv_sec  - tStart.tv_sec  - (tEnd.tv_nsec<tStart.tv_nsec?1:0);
	diff.tv_nsec = tEnd.tv_nsec - tStart.tv_nsec + (tEnd.tv_nsec<tStart.tv_nsec?1000000000:0);

	return ((double) diff.tv_sec) + ((double) diff.tv_nsec)/1e9;
}
#endif

int randState;

/** Predictable pseudo-random function to always generate the same output, regardless of running platform. */
int myRand()
{
	int const a = 1103515245;
	int const c = 12345;
	randState = a * randState + c;
	return (randState >> 16) & 0x7FFF;
}

void mySrand(int seed)
{
	randState = seed;
}
