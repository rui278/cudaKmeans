#include "hip/hip_runtime.h"
//
//  kernel.c
//  cudaKmeans
//
//  Created by Rui on 22/05/14.
//  Copyright (c) 2014 __Grupo215AAC__. All rights reserved.
//

#include <stdio.h>
#include <string.h>
#include <limits.h>

#include "kernel.h"

#include "hip/hip_runtime.h"
#include ""




/* Clears the vars cent_r, local_tot and tot */
__global__ void
	clearVars(
	int numCent,
	int dims,
	int numCentPerThread,
	data_t * cent_r,	/* Temporary accumulation cells for each centroid's mean coordinates. All the threads will accumulate here, so they must use atomicAdd. Length: dims * numCent. */
	int* tot			/* Number of total points of each centroid, summed from all CUDA blocks. */
	)
{
	int centOffset = (blockDim.x * blockIdx.x + threadIdx.x) * numCentPerThread;

	// Clear the shared variables for this block
	int c;
	for (c = 0; c < numCentPerThread; c++)
	{
		if (centOffset + c >= numCent)
			return;

		int j;
		for (j = 0; j < dims; j++)
		{
			cent_r[(centOffset + c) * dims + j] = 0;
		}

		// Clear the global variable of this centroid for all blocks
		tot[centOffset + c] = 0;
	}
}






/* Classify points to nearest centroid. Part 1 */
__global__ void
	classifyPoints(int NUMIT, int numPoints, int numCent, int dims,
	data_t * x, data_t * dist, data_t * cent,
	int * bestCent,
	data_t * cent_r,	/* Temporary accumulation cells for each centroid's mean coordinates. All the threads will accumulate here, so they must use atomicAdd. Length: dims * numCent. */
	int pointsPerThread,
	int* tot			/* Number of points of each centroid, from the points belonging to a certain CUDA block. */
	)
{
	int pointOffset = (blockDim.x * blockIdx.x + threadIdx.x) * pointsPerThread;

	int j, k;

	/* calculate distance matrix and minimum */
	data_t rMin;

	for (j=0; j<pointsPerThread; j++)
	{

		/* Last threads of last block may not have a round number of points */
		if (pointOffset + j >= numPoints)
		{
			return;
		}

		// Find this point's nearest centroid
		rMin=INT_MAX;
		for (k=0; k < numCent; k++)
		{
			int e;
			DIST_(j + pointOffset, k) = 0;
			for (e=0; e<dims; e++)
			{
				DIST_(j+pointOffset, k) += (X_(j+pointOffset, e) - CENT_(k, e)) * (X_(j+pointOffset, e) - CENT_(k, e));
			}
			
			if (DIST_(j+pointOffset, k) < rMin)
			{
				bestCent[j+pointOffset]=k;
				rMin=DIST_(j+pointOffset, k);
			}
		}

		int e;
		for (e = 0; e < dims; e++)
		{
			atomicAdd(&CENT_R_(bestCent[j+pointOffset], e), X_(j+pointOffset, e));
		}
		atomicAdd( &(tot[bestCent[j+pointOffset]]) , 1 );
	}
}







__global__ void
	calculateCentroids(int NUMIT, int numPoints, int numCent, int dims,
	data_t * x,
	data_t * dist,
	data_t * cent,
	data_t * mean,
	int * bestCent,
	data_t * cent_r,	/* Temporary accumulation cells for each centroid's mean coordinates. All the threads will accumulate here, so they must use atomicAdd. Length: dims * numCent. */
	int numCentPerThread,
	int* tot			/* Number of points of each centroid, from the points belonging to a certain CUDA block. */
	)
{
	int centOffset = (blockDim.x * blockIdx.x + threadIdx.x) * numCentPerThread;

	int k;
	for (k=0; k<numCentPerThread; k++)
	{
		if (centOffset + k >= numCent)
			return;

		/* If centroid has more than 0 points associated (normal), relocate it to mean of its points. */
		if (tot[k + centOffset] > 0)
		{
			int e;
			for (e=0; e<dims; e++)
				CENT_(k + centOffset, e)=CENT_R_(k + centOffset, e) / tot[k + centOffset];
		}
		/* Else, relocate it to the mean of the other centroids (put it near points) */
		else
		{
			int e;
			for (e=0; e<dims; e++)
				CENT_(k + centOffset, e)=mean[e];
		}
	}


}